#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MATRIX_SIZE 8192
#define CPU_MATRIX_SIZE 1024


__global__ void gpu_matrix_mult(int *a,int *b, int *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if( col < n && row < n)
    {
        int sum = 0;
        for(int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

void cpu_matrix_mult (int *a, int *b, int *c, int n)
{
    int i,j,k;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            int sum_mult = 0;
            for (k = 0; k < n; k++)
            {
                sum_mult += a[i*n+k] * b[k*n+j];
            }
            c[i*n+j] = sum_mult;
        }
    }
}

int main(int argc, char const *argv[])
{
    int block_size;

    /// retrieve some info about the CUDA device
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i);
      printf("  Device name: %s\n", prop.name);
      printf("  max Blocks Per MultiProcessor: %d\n", prop.maxBlocksPerMultiProcessor);
      printf("  max Threads Per MultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
      printf("  max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
      printf("  num SM: %d\n", prop.multiProcessorCount);
      printf("  num bytes sharedMem Per Block: %d\n", prop.sharedMemPerBlock);
      printf("  num bytes sharedMem Per Multiprocessor: %d\n", prop.sharedMemPerMultiprocessor);
      printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
      printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
    {
        int *a, *b, *c;
        a = (int*)malloc(sizeof(int)*CPU_MATRIX_SIZE*CPU_MATRIX_SIZE);
        b = (int*)malloc(sizeof(int)*CPU_MATRIX_SIZE*CPU_MATRIX_SIZE);
        c = (int*)malloc(sizeof(int)*CPU_MATRIX_SIZE*CPU_MATRIX_SIZE);
        // initialize matrix A
        for (int i = 0; i < CPU_MATRIX_SIZE; ++i) {
            for (int j = 0; j < CPU_MATRIX_SIZE; ++j) {
                a[i * CPU_MATRIX_SIZE + j] = 2;
            }
        }
        // initialize matrix B
        for (int i = 0; i < CPU_MATRIX_SIZE; ++i) {
            for (int j = 0; j < CPU_MATRIX_SIZE; ++j) {
                b[i * CPU_MATRIX_SIZE + j] = 3;
            }
        }
        // sequential version of matrix multiplication
        clock_t begin = clock();
        cpu_matrix_mult(a, b, c, CPU_MATRIX_SIZE);
        clock_t end = clock();
        double time_spent = ((double)((end - begin)) * 1000) / CLOCKS_PER_SEC;
        printf("Time elapsed on naive CPU sequential matrix multiplication of %dx%d . %dx%d: %f ms\n\n", CPU_MATRIX_SIZE, CPU_MATRIX_SIZE, CPU_MATRIX_SIZE, CPU_MATRIX_SIZE, time_spent);
        free(a);
        free(b);
        free(c);
    }

    for(block_size= 4; block_size <= 32; block_size *= 2)
    {
        int *a, *b, *c;
        hipMallocManaged((void **) &a, sizeof(int)*MATRIX_SIZE*MATRIX_SIZE);
        hipMallocManaged((void **) &b, sizeof(int)*MATRIX_SIZE*MATRIX_SIZE);
        hipMallocManaged((void **) &c, sizeof(int)*MATRIX_SIZE*MATRIX_SIZE);

        // initialize matrix A
        for (int i = 0; i < MATRIX_SIZE; ++i) {
            for (int j = 0; j < MATRIX_SIZE; ++j) {
                a[i * MATRIX_SIZE + j] = 2;
            }
        }

        // initialize matrix B
        for (int i = 0; i < MATRIX_SIZE; ++i) {
            for (int j = 0; j < MATRIX_SIZE; ++j) {
                b[i * MATRIX_SIZE + j] = 3;
            }
        }


        float  naive_gpu_elapsed_time_ms;

        // some events to count the execution time
        //clock_t st, end;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);


        unsigned int grid_rows = (MATRIX_SIZE + block_size - 1) / block_size;
        unsigned int grid_cols = (MATRIX_SIZE + block_size - 1) / block_size;
        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(block_size, block_size);


        hipEventRecord(start, 0);
        gpu_matrix_mult<<<dimGrid, dimBlock>>>(a, b, c, MATRIX_SIZE);
        hipDeviceSynchronize();

        // time counting terminate

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // compute time elapsed on GPU computing
        hipEventElapsedTime(&naive_gpu_elapsed_time_ms, start, stop);
        printf("Time elapsed on naive GPU matrix multiplication of %dx%d . %dx%d (%d): %f ms.\n\n", MATRIX_SIZE, MATRIX_SIZE, MATRIX_SIZE, MATRIX_SIZE, block_size, naive_gpu_elapsed_time_ms);


        // free memory
        hipFree(a);
        hipFree(b);
        hipFree(c);
    }

    return 0;
}