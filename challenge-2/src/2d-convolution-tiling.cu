#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <fstream>

#define BLOCK_WIDTH 32

using namespace std;

/**
 * 2D convolution kernel with tiling.
 * It optimizes the convolution operation by using shared memory to store the input matrix tile,
 * reducing the number of global memory accesses.
 *
 * The kernel first loads a tile of the input matrix into shared memory,
 * then performs the convolution operation within the tile.
 *
 * @param cuda_input_m Input matrix.
 * @param mask Mask matrix.
 * @param cuda_output_m Output matrix.
 * @param height Height of the input matrix.
 * @param width Width of the input matrix.
 * @param mask_width Width of the mask matrix.
 * @param N_TILE_WIDTH Tile width.
 */
__global__ void convolution_2D_tiled_kernel(
    const float* cuda_input_m,
    const float* __restrict__ mask,
    float* cuda_output_m,
    const size_t height,
    const size_t width,
    const size_t mask_width,
    const int N_TILE_WIDTH
) {
    // shared memory
    __shared__ float tile_shared_memory[BLOCK_WIDTH][BLOCK_WIDTH];

    // init
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int n_row = blockIdx.y * N_TILE_WIDTH + ty;
    const int n_col = blockIdx.x * N_TILE_WIDTH + tx;
    const int m_row = n_row - mask_width / 2;
    const int m_col = n_col - mask_width / 2;

    // boundary condition
    if(m_row >= 0 && m_row < height && m_col >= 0 && m_col < width) {
        // load element from input matrix to shared memory in the respective tile position
        tile_shared_memory[ty][tx] = cuda_input_m[m_row * width + m_col];
    } else {
        // avoid branch divergence
        tile_shared_memory[ty][tx] = 0;
    }

    // barrier synchronization
    __syncthreads();

    // boundary condition to avoid out-of-bounds access, because we calculate only N_TILE_LENGTH elements
    if(ty < N_TILE_WIDTH && tx < N_TILE_WIDTH && n_row < height && n_col < width)
    {
        // convolution result
        float convolution_result = 0;
        // calculate convolution result
        for(int i = 0; i < mask_width; ++i) {
            for(int j = 0; j < mask_width; ++j) {
                convolution_result += mask[i * mask_width + j] * tile_shared_memory[ty + i][tx + j];
            }
        }
        // save convolution result to output matrix (barrier synchronization not needed)
        cuda_output_m[n_row * width + n_col] = convolution_result;
    }
}


/**
 * Print a matrix.
 * @param matrix Matrix to print.
 * @param height Height of the matrix.
 * @param width Width of the matrix.
 */
void print_matrix(const float* matrix, const int height, const int width) {
    if (matrix == nullptr) {
        throw invalid_argument("Matrix cannot be null");
    }
    for (int i = 0; i < height * width; ++i) {
        printf("%f ", matrix[i]);
        if (i % width == width - 1) {
            printf("\n");
        }
    }
}

/**
 * Create a matrix with random values between 0 and upper_bound.
 * @param result Result matrix.
 * @param rows Number of rows of the output matrix.
 * @param cols Number of cols of the output matrix.
 * @param lower_bound Lower limit for random number generation.
 * @param upper_bound Upper limit for random number generation.
 * @throw invalid_argument If mask is null.
 */
void create_random_matrix(float *result, const int rows, const int cols, const int lower_bound, const int upper_bound) {
    if (lower_bound > upper_bound) {
        throw invalid_argument("Lower bound cannot be greater than upper bound");
    }
    if (result == nullptr) {
        throw invalid_argument("Result matrix cannot be null");
    }
    // init bound
    const int boundary = rows * cols;
    // insert values
    for (int i = 0; i < boundary; ++i)
        result[i] = ((random() % upper_bound) + lower_bound);
}

/**
 * Create a constant matrix with a specific value.
 * @param result Constant matrix result.
 * @param rows Rows of the final matrix, it should be square.
 * @param cols Cols of the final matrix, it should be square.
 * @param value Value to fill the matrix.
 * @throw invalid_argument If result is null.
 */
void create_constant_matrix(float *result, const int rows, const int cols, const float value) {
    if (result == nullptr)
        throw invalid_argument("Result matrix cannot be null");
    // init bound
    const int boundary = rows * cols;
    // insert values
    for (int i = 0; i < boundary; ++i)
        result[i] = value;
}

/**
 * Verify the result of the convolution operation using the CPU.
 * @param matrix Input matrix.
 * @param mask Mask matrix.
 * @param result Result matrix to verify.
 * @param height Height of the input matrix.
 * @param width Width of the input matrix.
 * @param mask_dim Dimension of the mask matrix.
 */
void verify_result(
    const float *matrix,
    const float *mask,
    const float *result,
    const int height,
    const int width,
    const int mask_dim
) {
    if (matrix == nullptr || mask == nullptr || result == nullptr) {
        throw invalid_argument("Matrix, mask, and result cannot be null");
    }
    const int mask_offset = mask_dim / 2;
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            float expected_convolution = 0.0;
            for (int k = 0; k < mask_dim; k++) {
                for (int l = 0; l < mask_dim; l++) {
                    const int r = i - mask_offset + k;
                    if (const int c = j - mask_offset + l; r >= 0 && r < height && c >= 0 && c < width) {
                        expected_convolution += matrix[r * width + c] * mask[k * mask_dim + l];
                    }
                }
            }
            const float convolution = result[i * width + j];
            assert(convolution == expected_convolution);
        }
    }
}


int main() {
    // init
    int mask_width = 0, matrix_width = 0, matrix_height = 0;
    int seed = 0, warmup = 0;

    // try to get env variable about the matrix size and mask size
    try {
        mask_width = stoi(getenv("MASK_SIZE"));
        matrix_width = stoi(getenv("MATRIX_WIDTH"));
        matrix_height = stoi(getenv("MATRIX_HEIGHT"));
        if (!(mask_width > 0 && matrix_width > 0 && matrix_height > 0 && mask_width % 2 != 0)) {
            throw invalid_argument("Invalid argument");
        }
    } catch (...) {
        printf("Error reading MASK_SIZE env variable; it must be an integer.\n");
        return 1;
    }
    // try to get env variable about the seed
    try {
        seed = stoi(getenv("SEED"));
    } catch (...) {
        printf("WARNING: SEED env variable not found; random values will be generated.\n\n");
    }
    // try to get env variable about the warmup
    try {
        warmup = stoi(getenv("WARMUP"));
    } catch (...) {
        printf("WARNING: WARMUP env variable not set, a single run will be performed.\n\n");
    }

    // retrieve some info about the CUDA device
    hipGetDeviceCount(nullptr);
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, 0);
    printf("Device Number: %d\n", 0);
    printf("  Device name: %s\n", prop.name);
    printf("  max Blocks Per MultiProcessor: %d\n", prop.maxBlocksPerMultiProcessor);
    printf("  max Threads Per MultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
    printf("  num SM: %d\n", prop.multiProcessorCount);
    printf("  num bytes sharedMem Per Block: %lu\n", prop.sharedMemPerBlock);
    printf("  num bytes sharedMem Per Multiprocessor: %lu\n", prop.sharedMemPerMultiprocessor);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);


    /** Execution **/
    // init
    float naive_gpu_elapsed_time_ms;
    const int N_TILE_WIDTH = BLOCK_WIDTH - (mask_width - 1);
    float* input_m = static_cast<float *>(malloc(matrix_height * matrix_width * sizeof(float)));
    float* mask = static_cast<float *>(malloc(mask_width * mask_width * sizeof(float)));
    float* output_m = static_cast<float *>(malloc(matrix_width * matrix_height * sizeof(float)));

    // populate
    if (seed != 0) {
        srand(seed);
    }
    create_random_matrix(input_m, matrix_height, matrix_width, 1, 100);
    create_random_matrix(mask, mask_width, mask_width, 1, 5);

    // =============================================== START CONVOLUTION ===============================================
    // time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // define grid and block dimensions
    dim3 dim_grid(
        ceil(matrix_width / static_cast<float>(N_TILE_WIDTH)),
        ceil(matrix_height / static_cast<float>(N_TILE_WIDTH))
    );
    dim3 dim_block(BLOCK_WIDTH, BLOCK_WIDTH);
    const size_t bytes_input_m = matrix_height * matrix_width * sizeof(int);
    const size_t bytes_mask = mask_width * mask_width * sizeof(int);

    // allocate memory in the device
    float* cuda_input_m;
    float* cuda_mask;
    float* cuda_output_m;
    hipMalloc(reinterpret_cast<void **>(&cuda_input_m), bytes_input_m);
    hipMalloc(reinterpret_cast<void **>(&cuda_mask), bytes_mask);
    hipMalloc(reinterpret_cast<void **>(&cuda_output_m), bytes_input_m); // same bytes as input

    // initialize memory in the device
    hipMemcpy(cuda_input_m, input_m, bytes_input_m, hipMemcpyHostToDevice);
    hipMemcpy(cuda_mask, mask, bytes_mask, hipMemcpyHostToDevice);

    // warmup
    for (int i = 0; i < warmup; ++i) {
        convolution_2D_tiled_kernel<<<dim_grid, dim_block>>>(
            cuda_input_m, cuda_mask, cuda_output_m, matrix_height, matrix_width, mask_width, N_TILE_WIDTH
        );
    }

    hipEventRecord(start, nullptr);
    convolution_2D_tiled_kernel<<<dim_grid, dim_block>>>(
        cuda_input_m, cuda_mask, cuda_output_m, matrix_height, matrix_width, mask_width, N_TILE_WIDTH
    );
    hipDeviceSynchronize();
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);

    hipMemcpy(output_m, cuda_output_m, bytes_input_m, hipMemcpyDeviceToHost);


    hipEventElapsedTime(&naive_gpu_elapsed_time_ms, start, stop);

    printf("Check the result of the convolution operation using the CPU...\n");
    verify_result(input_m, mask, output_m, matrix_height, matrix_width, mask_width);
    printf("Verification passed!\n");

    // debug: print the output matrix
    // print_matrix(output_m, matrix_height, matrix_width);
    printf("Time elapsed on naive GPU 2D-convolution of %dx%d (block %d): %f ms.\n\n",
        matrix_height, matrix_width, BLOCK_WIDTH, naive_gpu_elapsed_time_ms);

    hipFree(cuda_input_m);
    hipFree(cuda_mask);
    hipFree(cuda_output_m);

    // ================================================ END CONVOLUTION ================================================

    // free
    free(input_m);
    free(mask);
    free(output_m);
    return 0;
}
