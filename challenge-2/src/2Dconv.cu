#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
void check_cuda_error(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error (" << msg << "): " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

__global__ void convolution_2d(int* input, int* output, int height, int width, int* mask, int mask_dim) {
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    const int mask_radius = mask_dim / 2;

    if (row < height && col < width) {
        int result = 0;

        for (int i = -mask_radius; i <= mask_radius; i++) {
            for (int j = -mask_radius; j <= mask_radius; j++) {
                int cur_row = row + i;
                int cur_col = col + j;

                if (cur_row >= 0 && cur_row < height && cur_col >= 0 && cur_col < width) {
                    result += input[cur_row * width + cur_col] * mask[(i + mask_radius) * mask_dim + (j + mask_radius)];
                }
            }
        }

        output[row * width + col] = result;
    }
}


// Funzione per la convoluzione 2D sulla CPU
void convolution_2d_cpu(int *matrix, int *result, int height, int width, int *mask, int mask_dim) {
    int mask_offset = mask_dim / 2;
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            int temp = 0;
            for (int k = 0; k < mask_dim; k++) {
                for (int l = 0; l < mask_dim; l++) {
                    int r = i - mask_offset + k;
                    int c = j - mask_offset + l;
                    if (r >= 0 && r < height && c >= 0 && c < width) {
                        temp += matrix[r * width + c] * mask[k * mask_dim + l];
                    }
                }
            }
            result[i * width + j] = temp;
        }
    }
}

void initialize_matrix(int* matrix, int height, int width, int value) {
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            matrix[i * width + j] = value;
        }
    }
}

void initialize_mask(int* mask, int mask_dim, int value) {
    for (int i = 0; i < mask_dim; i++) {
        for (int j = 0; j < mask_dim; j++) {
            mask[i * mask_dim + j] = value;
        }
    }
}


// Funzione per verificare il risultato
void verify_result(int *matrix, int *mask, int *result, int height, int width, int mask_dim) {
    int mask_offset = mask_dim / 2;
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            int temp = 0;
            for (int k = 0; k < mask_dim; k++) {
                for (int l = 0; l < mask_dim; l++) {
                    int r = i - mask_offset + k;
                    int c = j - mask_offset + l;
                    if (r >= 0 && r < height && c >= 0 && c < width) {
                        temp += matrix[r * width + c] * mask[k * mask_dim + l];
                    }
                }
            }
            assert(result[i * width + j] == temp);
        }
    }
}

// print a matrix
void print_matrix(const int *matrix, int rows, int cols, const std::string &label) {
    std::cout << label << " (" << rows << "x" << cols << "):" << std::endl;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            std::cout << matrix[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <matrix_height> <matrix_width> <mask_dim>\n";
        return 1;
    }

    int height = std::atoi(argv[1]);
    int width = std::atoi(argv[2]);
    int mask_dim = std::atoi(argv[3]);

    if (height <= 0 || width <= 0 || mask_dim <= 0 || mask_dim % 2 == 0) {
        std::cerr << "Error: Dimensions must be positive and mask_dim must be odd.\n";
        return 1;
    }

    size_t matrix_size = height * width * sizeof(int);
    size_t mask_size = mask_dim * mask_dim * sizeof(int);

    int* h_input = new int[height * width];
    int* h_output = new int[height * width];
    int* h_mask = new int[mask_dim * mask_dim];

    initialize_matrix(h_input, height, width, 1);
    initialize_mask(h_mask, mask_dim, 1);

    int *d_input, *d_output, *d_mask;
    check_cuda_error(hipMalloc(&d_input, matrix_size), "hipMalloc d_input");
    check_cuda_error(hipMalloc(&d_output, matrix_size), "hipMalloc d_output");
    check_cuda_error(hipMalloc(&d_mask, mask_size), "hipMalloc d_mask");

    check_cuda_error(hipMemcpy(d_input, h_input, matrix_size, hipMemcpyHostToDevice), "hipMemcpy d_input");
    check_cuda_error(hipMemcpy(d_mask, h_mask, mask_size, hipMemcpyHostToDevice), "hipMemcpy d_mask");

    dim3 block_dim(16, 16);
    dim3 grid_dim((width + block_dim.x - 1) / block_dim.x, (height + block_dim.y - 1) / block_dim.y);

      // No tiling version
    auto start_no_tiling = std::chrono::high_resolution_clock::now();
    convolution_2d<<<grid_dim, block_dim>>>(d_input, d_output, height, width, d_mask, mask_dim);
    check_cuda_error(hipDeviceSynchronize(), "Kernel execution");
    auto end_no_tiling = std::chrono::high_resolution_clock::now();

    check_cuda_error(hipMemcpy(h_output, d_output, matrix_size, hipMemcpyDeviceToHost), "hipMemcpy h_output");
    verify_result(h_input, h_mask, h_output, height, width, mask_dim);
     std::cout << "VERIFY COMPLETED SUCCESSFULLY!\n";
    //std::cout << "Output Matrix:\n";
    //for (int i = 0; i < height; i++) {
       // for (int j = 0; j < width; j++) {
         //   std::cout << h_output[i * width + j] << " ";
       // }
        //std::cout << "\n";
    //}
    //print_matrix(h_output, height, width, "result on d2conv on GPU");

      // CPU version
    int* result_cpu = new int[height * width];
    auto start_cpu = std::chrono::high_resolution_clock::now();
    convolution_2d_cpu(h_input, result_cpu, height, width, h_mask, mask_dim);
    auto end_cpu = std::chrono::high_resolution_clock::now();
   // print_matrix(result_cpu, height, width, "result on 2dconv on CPU");



 std::cout << "Time (GPU No Tiling): "
              << std::chrono::duration_cast<std::chrono::milliseconds>(end_no_tiling - start_no_tiling).count()
              << " ms\n";

    std::cout << "Time (CPU): "
              << std::chrono::duration_cast<std::chrono::milliseconds>(end_cpu - start_cpu).count()
              << " ms\n";


    delete[] h_input;
    delete[] h_output;
    delete[] h_mask;
    check_cuda_error(hipFree(d_input), "hipFree d_input");
    check_cuda_error(hipFree(d_output), "hipFree d_output");
    check_cuda_error(hipFree(d_mask), "hipFree d_mask");

    return 0;
}